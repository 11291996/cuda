//error handling


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h> 

#ifndef CHECK_ERROR_H //this is to avoid multiple definitions
#define CHECK_ERROR_H
//returns the CUDA error string associated with the given error code
#define chkErr(ans) { gpuAssert((ans), __FILE__, __LINE__); }
//this is the function that actually does the error checking
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
        if (code != hipSuccess) {
            fprintf(stderr, "ChkErr: %s %s line %d\n", hipGetErrorString(code), file, line);
            if (abort) exit(code);
        }
}

#endif //CHECK_ERROR_H //close the ifndef

int main() {
    int *d_a;
    chkErr(hipMalloc(&d_a, 9000000000000000000)); //this will return an error
}